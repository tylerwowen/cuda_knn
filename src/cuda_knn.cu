#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cuda_knn.cu
 Author      : Tyler Ouyang
 Version     :
 Copyright   : Copyright © 2016 Tyler Ouyang. All rights reserved.
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <stdlib.h>

#include "common.cuh"
#include "common.h"
#include "utils.cuh"
#include "thrust_utils.h"

using namespace std;

__device__
float calculateCOSDistance(
    Rating *r1Start,
    Rating *r1End,
    Rating *r2Start,
    Rating *r2End) {

  float dotProduct = 0.0, r1NormSQ = 0.0, r2NormSQ = 0.0;

  while (r1Start < r1End && r2Start < r2End) {
    if (r1Start->x > r2Start->x) {
      // treat r1Start->rating as 0
      r2NormSQ += r2Start->y * r2Start->y;
      r2Start++;
    } else if (r1Start->x == r2Start->x) {
      dotProduct += r1Start->y * r2Start->y;
      r1NormSQ += r1Start->y * r1Start->y;
      r2NormSQ += r2Start->y * r2Start->y;
      r1Start++;
      r2Start++;
    } else {
      // treat r2Start->y as 0
      r1NormSQ += r1Start->y * r1Start->y;
      r1Start++;
    }
  }
  // finish baseUser tail, if any
  while (r1Start < r1End) {
    r1NormSQ += r1Start->y * r1Start->y;
    r1Start++;
  }
  // finish neighbor tail, if any
  while (r2Start < r2End) {
    r2NormSQ += r2Start->y * r2Start->y;
    r2Start++;
  }
  // distance
  return dotProduct / (sqrt(r1NormSQ) * sqrt(r2NormSQ));
}

/**
 * CUDA kernel that computes distances between every two users in d_trainUsers
 */
__global__
void calculateAllDistance(
    int stageStartUser,
    int numUsers,
    int *d_trainUsers,
    Rating *allRatings,
    float *d_distances) {

  int globalUserId = stageStartUser + blockIdx.x * blockDim.x + threadIdx.x;
  // user id relative in stage
  int localUserId = blockIdx.x * blockDim.x + threadIdx.x;
  // TODO: experimental, need optimization
  // space for TILE_SIZE * 2 users, each one has at most TILE_DEPTH ratings
  __shared__ Rating ratings[TILE_DEPTH * TILE_SIZE * 2];

  int baseNumRatings = d_trainUsers[globalUserId];
  int numRatings = min(baseNumRatings, TILE_DEPTH);
  Rating *baseStart = ratings + (threadIdx.x + TILE_SIZE) * TILE_DEPTH,
      *baseEnd = baseStart + numRatings;

  // copy data to shared memory, base users are the last TILE_SIZE users in ratings
  Rating *copyFrom = allRatings + globalUserId * TILE_DEPTH;
#pragma unroll
  for (int i = threadIdx.y; i < numRatings; i += TILE_SIZE)
    baseStart[i] = copyFrom[i];
  __syncthreads();

  // TILE_SIZE user per iteration for now
  for (int i = threadIdx.y; i < numUsers; i += TILE_SIZE) {
    int neighborNumRatings = d_trainUsers[i];
    int nbNumRatings = min(neighborNumRatings, TILE_DEPTH);
    Rating *neighborStart = ratings + threadIdx.y * TILE_DEPTH,
        *neighborEnd = neighborStart + nbNumRatings;

    copyFrom = allRatings + i * TILE_DEPTH;
    // copy data to shared memory, neighbors are the first TILE_SIZE users
#pragma unroll
    for (int j = threadIdx.x; j < nbNumRatings; j += TILE_SIZE)
      neighborStart[j] = copyFrom[j];
    __syncthreads();

    d_distances[localUserId * numUsers + i]
                = calculateCOSDistance(baseStart, baseEnd, neighborStart, neighborEnd);

//    if (globalUserId == 766) {
//      printf("distance from user %d to user %d is %.20lf\n", globalUserId, i,
//          d_distances[localUserId * numUsers + i]);
//    }
    __syncthreads();
  }

}

/**
 * CUDA kernel that computes KNN
 */
__global__
void knn(int numUsers, int k,
    short *idxIdMap,
    Rating *trainRatings, Rating *testRatings, int testRatingOffset,
    int *trainUser,
    int *ratingSums, int *ratingCounts) {

  extern __shared__ Rating sharedRatings[];
  // space to store ratings found by each thread
  short *foundRatings = (short*) &sharedRatings[TILE_DEPTH * blockDim.y];
  short *finished = (short*) &foundRatings[blockDim.x * blockDim.y];

  int threadId = threadIdx.x * blockDim.y + threadIdx.y;
  // initialize shared memory
  foundRatings[threadId] = 0;
  if (threadIdx.y == 0 ) finished[threadIdx.x] = 0;

  int sumOfRatings = 0;
  int numOfMatchedNeighbors = 0;

  int testItemId = testRatings[testRatingOffset + threadIdx.x].x;

  // TODO: consider stopping at 20*K instead of numUsers
  for (int neighborIdx = threadIdx.y; neighborIdx < numUsers; neighborIdx += blockDim.y) {
    // load ratings of blockDim.y users to shared memory
    int nbNumRatings = trainUser[idxIdMap[neighborIdx]];
    nbNumRatings = min(nbNumRatings, TILE_DEPTH);

    Rating *neighborStart = sharedRatings + threadIdx.y * TILE_DEPTH;
    Rating *copyFrom = trainRatings + idxIdMap[neighborIdx] * TILE_DEPTH;

    // TODO: optimize loading by using row major access
    for (int j = threadIdx.x; j < nbNumRatings; j += blockDim.x)
      neighborStart[j] = copyFrom[j];
    __syncthreads();

    if (!finished[threadIdx.x]) {
      foundRatings[threadId] = findItemRating(testItemId, neighborStart, nbNumRatings);
      __syncthreads();

      // thread 0 of each movie collects information
      if (threadIdx.y == 0) {
        for (int i = 0; i < blockDim.y; i++) {
          if (numOfMatchedNeighbors == k) {
            finished[threadIdx.x] = 1;
            break;
          }
          int rate = foundRatings[threadId + i];
          if (rate > 0) {
            sumOfRatings += rate;
            numOfMatchedNeighbors++;
          }
        }
      }
    }
  }
  if (threadIdx.y == 0) {
    ratingSums[threadIdx.x] = sumOfRatings;
    ratingCounts[threadIdx.x] = numOfMatchedNeighbors;
//    printf("prediction for item %d is %f\n", testItemId, (float)sumOfRatings/numOfMatchedNeighbors/2);
  }
}

void moveRatingsToDevice(
    H_Users h_trainUsers,
    int **d_users,
    Rating **d_ratings) {

  // make numTrainUsers a multiple of TILE_SIZE
  int numUsers = h_trainUsers.size() / TILE_SIZE * TILE_SIZE;
  int numRatings = numUsers * TILE_DEPTH;
  int *h_users = new int[numUsers];

  for (int i = 0; i < numUsers; i++)
    h_users[i] = 0;

  Rating *h_ratings = new Rating[sizeof(Rating) * numRatings];
  checkCudaErrors(hipMalloc((void **) d_ratings, sizeof(Rating) * numRatings));
  cout << "size of train ratings in bytes: " << sizeof(Rating) * numRatings << endl;

  for (int i = 0; i < numUsers; i++) {
    int numRatings = h_trainUsers[i].size();

    // copy vector to intermediate host array
    for (int j = 0; j < numRatings; j++) {
      h_ratings[i * TILE_DEPTH + j].x = h_trainUsers[i][j].first;
      h_ratings[i * TILE_DEPTH + j].y = h_trainUsers[i][j].second * 2;
    }

    h_users[i] = numRatings;
  }
  // move data from host to device
  checkCudaErrors(hipMemcpy(*d_ratings, h_ratings, sizeof(Rating) * numRatings, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **) d_users, sizeof(int) * numUsers));
  checkCudaErrors(hipMemcpy(*d_users, h_users, sizeof(int) * numUsers, hipMemcpyHostToDevice));

  delete[] h_ratings;
  delete[] h_users;
}

void initUsers(User *users, int num) {
  for (int i = 0; i < num; i++)
    users[i] = {NULL, 0};
  }

void moveTestRatingsToDevice(
    H_Users h_testUsers,
    User *h_users,
    Rating **d_ratings,
    int numUsers,
    int testUserRatingCount) {

  initUsers(h_users, numUsers);
  numUsers = min(numUsers, (int) h_testUsers.size());

  Rating *h_ratings = new Rating[sizeof(Rating) * testUserRatingCount];
  checkCudaErrors(hipMalloc((void **) d_ratings, sizeof(Rating) * testUserRatingCount));

  int ratingsSoFar = 0;
  for (int i = 0; i < numUsers; i++) {
    int numRatings = h_testUsers[i].size();
    if (numRatings < 1) continue;

    // copy vector to intermediate host array
    for (int j = 0; j < numRatings; j++) {
      h_ratings[ratingsSoFar + j].x = h_testUsers[i][j].first;
      h_ratings[ratingsSoFar + j].y = h_testUsers[i][j].second * 2;
    }
    // save index
    h_users[i].ratings = *d_ratings + ratingsSoFar;
    h_users[i].numRatings = numRatings;

    ratingsSoFar += numRatings;
  }
  // move data from host to device
  checkCudaErrors(hipMemcpy(*d_ratings, h_ratings, sizeof(Rating) * testUserRatingCount, hipMemcpyHostToDevice));

  delete[] h_ratings;
}

void cudaCore(
    int trainUserRatingCount,
    int testUserRatingCount,
    H_Users h_trainUsers,
    H_Users h_testUsers,
    int k) {

  int *d_trainUsers, *d_ratingSums, *d_ratingCounts;
  int h_ratingCounts[CONC_ITEMS_NUM] = { 0 }, h_ratingSums[CONC_ITEMS_NUM] = { 0 };
  Rating *d_trainRatings, *d_testRatings;
  int numTrainUsers = h_trainUsers.size() / TILE_SIZE * TILE_SIZE;
  User *h_testUsersIdx = new User[numTrainUsers];
  float *d_distances;
  short *d_idxIdMap;

  int predictedCount = 0;
  double errorSum = 0, errorSumSq = 0;

  cout << "trainUserRatingCount: " << trainUserRatingCount << endl;
  cout << "number of users: " << h_trainUsers.size() << "; effective user: " << numTrainUsers << endl;
  cout << "testUserRatingCount: " << testUserRatingCount << endl;
  cout << "number of test users: " << h_testUsers.size() << endl;

  moveRatingsToDevice(h_trainUsers, &d_trainUsers, &d_trainRatings);
  moveTestRatingsToDevice(h_testUsers, h_testUsersIdx, &d_testRatings, numTrainUsers, testUserRatingCount);
  cout << "data moved to device\n";

  // get free memory
  size_t freeMemSize, totalMemSize;
  checkCudaErrors(hipMemGetInfo(&freeMemSize, &totalMemSize));
  cout << "device has " << freeMemSize << " free global memory\n";

  checkCudaErrors(hipMalloc((void **) &d_ratingSums, CONC_ITEMS_NUM * sizeof(int)));
  checkCudaErrors(hipMalloc((void **) &d_ratingCounts, CONC_ITEMS_NUM * sizeof(int)));
  checkCudaErrors(hipMalloc((void **) &d_idxIdMap, numTrainUsers * sizeof(short)));

  // calculate how many distances GPU can store, e.g. size of stage
  size_t ratingsSize = numTrainUsers * TILE_DEPTH * sizeof(Rating);
  freeMemSize -= ratingsSize * 10;
  cout << "train rating size " << ratingsSize << "\nfreeMemSize is " << freeMemSize << endl;
  int stageHeight = min(freeMemSize / (numTrainUsers * sizeof(float)) / TILE_SIZE, (long) numTrainUsers / TILE_SIZE);

  // allocate memory for distances
  checkCudaErrors(hipMalloc((void **) &d_distances, sizeof(float) * numTrainUsers * stageHeight * TILE_SIZE));
  hipDeviceSynchronize();

  dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
  cout << "each kernel has " << stageHeight << " blocks\n";
  cout << (numTrainUsers + stageHeight * TILE_SIZE - 1) / (stageHeight * TILE_SIZE) << " kernels will be launched\n";

  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  for (int stageStartUser = 0; stageStartUser < numTrainUsers; stageStartUser += stageHeight * TILE_SIZE) {
    int effectiveStageHeight = min(stageHeight, (numTrainUsers - stageStartUser) / TILE_SIZE);
    calculateAllDistance<<<effectiveStageHeight, threadsPerBlock>>>
    (stageStartUser, numTrainUsers, d_trainUsers, d_trainRatings, d_distances);

    // KNN
    for (int testUserIdOffset = 0; testUserIdOffset < effectiveStageHeight * TILE_SIZE; testUserIdOffset++) {
      int numTestItems = h_testUsersIdx[stageStartUser + testUserIdOffset].numRatings;
      if (numTestItems < 1) continue;

      // sort
      sortNeighbors(d_distances + testUserIdOffset * numTrainUsers, numTrainUsers, d_idxIdMap);

      // predict
      int numBlocks = (numTestItems + CONC_ITEMS_NUM - 1) / CONC_ITEMS_NUM;
      int remaining = numTestItems;
      for (int block = 0; block < numBlocks; block++) {
        int itemsInBlock = min(remaining, CONC_ITEMS_NUM);
        remaining -= CONC_ITEMS_NUM;
        dim3 threadsPerBlock(itemsInBlock, NUM_NEIGHBORS);

        knn<<<1, threadsPerBlock, NUM_NEIGHBORS*TILE_DEPTH*sizeof(Rating) + (itemsInBlock*(NUM_NEIGHBORS+1))*sizeof(short)>>>
        (numTrainUsers, k,
            d_idxIdMap,
            d_trainRatings, h_testUsersIdx[stageStartUser + testUserIdOffset].ratings, block * CONC_ITEMS_NUM,
            d_trainUsers,
            d_ratingSums, d_ratingCounts);

        checkCudaErrors(hipMemcpy(h_ratingSums, d_ratingSums, sizeof(int) * itemsInBlock, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_ratingCounts, d_ratingCounts, sizeof(int) * itemsInBlock, hipMemcpyDeviceToHost));

        for (int i = 0; i < itemsInBlock; i++) {
          float actual = h_testUsers[stageStartUser + testUserIdOffset][i + block * CONC_ITEMS_NUM].second;
          if (h_ratingCounts[i] == 0)
            continue;
          float prediction = h_ratingSums[i] / (float) h_ratingCounts[i] / 2;
//          cout << "user: " << stageStartUser + testUserIdOffset + 1
//              << " item: " << h_testUsers[stageStartUser + testUserIdOffset][i+block * itemsInBlock].first
//              << " actual = " << actual << " predicted = "<< prediction << "\n";// " based on " << h_ratingCounts[i] << " ratings\n";
//          cout  << stageStartUser + testUserIdOffset + 1
//                        << ", " << h_testUsers[stageStartUser + testUserIdOffset][i+block * CONC_ITEMS_NUM].first
//                        << ", " << actual << ", "<< prediction << "\n";
          errorSum += fabs(actual - prediction);
          errorSumSq += pow(actual - prediction, 2);
          predictedCount++;
        }
      }
    }
    cout << "\nerror sum so far: " << errorSum << ", error sum squared so far " << errorSumSq << endl;
    double mae = errorSum / predictedCount,
        rmse = sqrt(errorSumSq / predictedCount);
    cout << "MAE = " << mae << endl;
    cout << "RMSE = " << rmse << endl;
    cout << "Predicted count so far = " << predictedCount << endl;
  }
//  printptr<<<1,1>>>(d_idxIdMap, numTrainUsers);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << "kernel ended, took " << milliseconds << "ms\n";

  double mae = errorSum / predictedCount,
    rmse = sqrt(errorSumSq / predictedCount);
  cout << "MAE = " << mae << endl;
  cout << "RMSE = " << rmse << endl;
  cout << "Predicted count = " << predictedCount << endl;

  hipDeviceSynchronize();
  /* Free memory */
  hipEventDestroy(start);
  hipEventDestroy(stop);
  checkCudaErrors(hipFree(d_trainRatings));
  checkCudaErrors(hipFree(d_testRatings));
  checkCudaErrors(hipFree(d_trainUsers));
  checkCudaErrors(hipFree(d_distances));
  checkCudaErrors(hipFree(d_ratingSums));
  checkCudaErrors(hipFree(d_ratingCounts));
  checkCudaErrors(hipFree(d_idxIdMap));
  hipDeviceReset();
  delete[] h_testUsersIdx;
}
