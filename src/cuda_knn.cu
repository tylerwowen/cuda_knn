#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cuda_knn.cu
 Author      : Tyler Ouyang
 Version     :
 Copyright   : Copyright © 2016 Tyler Ouyang. All rights reserved.
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <stdlib.h>

#include "common.cuh"
#include "common.h"
#include "thrust_utils.h"

using namespace std;

#define TILE_SIZE 24
#define TILE_DEPTH 256

__device__
float calculateCOSDistance(
    Rating *r1Start,
    Rating *r1End,
    Rating *r2Start,
    Rating *r2End) {

  float dotProduct = 0.0, r1NormSQ = 0.0, r2NormSQ = 0.0;

  while (r1Start < r1End && r2Start < r2End) {
    if (r1Start->x > r2Start->x) {
      // treat r1Start->rating as 0
      r2NormSQ += r2Start->y * r2Start->y;
      r2Start++;
    } else if (r1Start->x == r2Start->x) {
      dotProduct += r1Start->y * r2Start->y;
      r1NormSQ += r1Start->y * r1Start->y;
      r2NormSQ += r2Start->y * r2Start->y;
      r1Start++;
      r2Start++;
    } else {
      // treat r2Start->y as 0
      r1NormSQ += r1Start->y * r1Start->y;
      r1Start++;
    }
  }
  // finish baseUser tail, if any
  while (r1Start < r1End) {
    r1NormSQ += r1Start->y * r1Start->y;
    r1Start++;
  }
  // finish neighbor tail, if any
  while (r2Start < r2End) {
    r2NormSQ += r2Start->y * r2Start->y;
    r2Start++;
  }
  // distance
  return dotProduct / (sqrt(r1NormSQ) * sqrt(r2NormSQ));
}

/**
 * CUDA kernel that computes distances between every two users in d_users
 */
__global__
void calculateAllDistance(
    int stageStartUser,
    int numUsers,
    int *d_users,
    Rating *allRatings,
    float *d_distances) {

  int globalUserId = stageStartUser + blockIdx.x * blockDim.x + threadIdx.x;
  // user id in stage
  int localUserId = blockIdx.x * blockDim.x + threadIdx.x;
  // TODO: experimental, need optimization
  // space for TILE_SIZE * 2 users, each one has at most TILE_DEPTH ratings
  __shared__ Rating ratings[TILE_DEPTH * TILE_SIZE * 2];

  int baseNumRatings = d_users[globalUserId];
  int numRatings = min(baseNumRatings, TILE_DEPTH);
  Rating *baseStart = ratings + (threadIdx.x + TILE_SIZE) * TILE_DEPTH, *baseEnd = baseStart + numRatings;

  // copy data to shared memory, base users are the last TILE_SIZE users in ratings
  Rating *copyFrom = allRatings + globalUserId * TILE_DEPTH;
#pragma unroll
  for (int i = threadIdx.y; i < numRatings; i += TILE_SIZE)
    baseStart[i] = copyFrom[i];
  __syncthreads();

//  printf("hello from block %d thread x %d, thread y %d\n", blockIdx.x, threadIdx.x, threadIdx.y);

  int *tileStartUser = d_users;
  // TILE_SIZE user per time for now
  for (int i = 0; i < numUsers; i += TILE_SIZE, tileStartUser += TILE_SIZE) {
    int neighborNumRatings = tileStartUser[threadIdx.y];
    int nbNumRatings = min(neighborNumRatings, TILE_DEPTH);
    Rating *neighborStart = ratings + threadIdx.y * TILE_DEPTH, *neighborEnd = neighborStart + nbNumRatings;

    copyFrom = allRatings + (i + threadIdx.y) * TILE_DEPTH;
    // copy data to shared memory, neighbors are the first TILE_SIZE users
#pragma unroll
    for (int j = threadIdx.x; j < nbNumRatings; j += TILE_SIZE)
      neighborStart[j] = copyFrom[j];
    // TODO: what if there are more than TILE_DEPTH users
    __syncthreads();

    d_distances[localUserId * numUsers + i + threadIdx.y] = calculateCOSDistance(baseStart, baseEnd, neighborStart,
        neighborEnd);
    __syncthreads();
  }

  printf("distance from user %d to user %d is %.20lf\n", localUserId, threadIdx.y,
      d_distances[localUserId * numUsers + threadIdx.y]);
}

/**
 * CUDA kernel that computes KNN
 */
__global__
void knn() {

}

void moveRatingsToDevice(
    H_Users h_trainUsers,
    int **d_users,
    Rating **d_ratings) {

  // make numTrainUsers a multiple of TILE_SIZE
  int numTrainUsers = h_trainUsers.size() / TILE_SIZE * TILE_SIZE;
  int numRatings = numTrainUsers * TILE_DEPTH;
  int *h_users = new int[numTrainUsers];
  ;
  for (int i = 0; i < numTrainUsers; i++)
    h_users[i] = 0;

  Rating *h_ratings = new Rating[sizeof(Rating) * numRatings];
  checkCudaErrors(hipMalloc((void **) d_ratings, sizeof(Rating) * numRatings));
  cout << "total size of ratings in bytes: " << sizeof(Rating) * numRatings << endl;

  for (int i = 0; i < numTrainUsers; i++) {
    int numRatings = h_trainUsers[i].size();

    // copy vector to intermediate host array
    for (int j = 0; j < numRatings; j++) {
      h_ratings[i * TILE_DEPTH + j].x = h_trainUsers[i][j].first;
      h_ratings[i * TILE_DEPTH + j].y = h_trainUsers[i][j].second * 2;
    }

    h_users[i] = numRatings;

  }
  // move data from host to device
  checkCudaErrors(hipMemcpy(*d_ratings, h_ratings, sizeof(Rating) * numRatings, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **) d_users, sizeof(int) * numTrainUsers));
  checkCudaErrors(hipMemcpy(*d_users, h_users, sizeof(int) * numTrainUsers, hipMemcpyHostToDevice));

  delete[] h_ratings;
  delete[] h_users;
}

void computeAllDistances(
    int trainUserRatingCount,
    int testUserRatingCount,
    H_Users h_trainUsers,
    H_Users h_testUsers) {

  int *d_users;
  Rating *d_allRatings;
  int numTrainUsers = h_trainUsers.size() / TILE_SIZE * TILE_SIZE;
  float *d_distances;
  short *d_indIdMap;

  cout << "trainUserRatingCount: " << trainUserRatingCount << endl;
  cout << "number of users: " << h_trainUsers.size() << "; effective user: " << numTrainUsers << endl;

  moveRatingsToDevice(h_trainUsers, &d_users, &d_allRatings);
  cout << "data moved to device\n";

  // get global memory
  struct hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, 0));
  cout << "device has " << prop.totalGlobalMem << " global memory\n";

  // calculate how many distances GPU can store, e.g. size of stage
  int ratingsSize = numTrainUsers * TILE_DEPTH * sizeof(Rating);
  int freeMemSize = prop.totalGlobalMem - ratingsSize * 1.5;
  int stageHeight = min(freeMemSize / (numTrainUsers * sizeof(float)) / TILE_SIZE, (long) numTrainUsers / TILE_SIZE);

  // allocate memory for distances
  checkCudaErrors(hipMalloc((void **) &d_distances, sizeof(float) * numTrainUsers * stageHeight * TILE_SIZE));
  // allocate memory for map(neighborIndex->neighborUserId)
  checkCudaErrors(hipMalloc((void **) &d_indIdMap, sizeof(unsigned short) * numTrainUsers));

  dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
  cout << "each kernel has " << stageHeight << " blocks\n";
  cout << (numTrainUsers + stageHeight * TILE_SIZE - 1) / (stageHeight * TILE_SIZE) << " kernels will be launched\n";

  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  for (int stageStartUser = 0; stageStartUser < numTrainUsers; stageStartUser += stageHeight * TILE_SIZE) {
    int effectiveStageHeight = min(stageHeight, (numTrainUsers - stageStartUser) / TILE_SIZE);
    calculateAllDistance<<<effectiveStageHeight, threadsPerBlock>>>
    (stageStartUser, numTrainUsers, d_users, d_allRatings, d_distances);

    // KNN
    for (int userNum = 0; userNum < stageHeight * TILE_SIZE; userNum++) {
      // sort

      sortNeighbors(d_distances + userNum * numTrainUsers, numTrainUsers, &d_indIdMap);

    }
  }
  for (int x = 0; x < numTrainUsers; x++) {
    cout << d_indIdMap[x] << " ";
  }
  cout << endl;

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << "kernel ended, took " << milliseconds << "ms\n";

  /* Free memory */
  hipEventDestroy(start);
  hipEventDestroy(stop);
  checkCudaErrors(hipFree(d_allRatings));
  checkCudaErrors(hipFree(d_users));
  hipDeviceReset();
}

